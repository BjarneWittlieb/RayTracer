#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "MainDef.h"

#include "HittableList.h"
#include "Sphere.h"
#include "Camera.h"
#include "Vector3.h"
#include "Material.h"

#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <thread>

using namespace std;

// To which file the image will be written
const char* FILEPATH = "reflectionpic.ppm";
const int NUMBER_OF_THREADS = 1;

// Window measurments
const int WIDTH = 40;
const int HEIGHT = 20;
const double ASPECT_RATIO = double(WIDTH) / HEIGHT;

// Samples per pixel in Antiailising
const int SAMPLES_PER_PIXEL_SQRT = 10;
// The maximum bounds of a ray
const int MAX_DEPTH = 50;

// Wich Gamma value to use in the gamma correction.
const double GAMMA_VALUE = 2.0;


Vector3 RayColor(const Ray& r, const Hittable& world, int depth);

HittableList RandomScene() {
	HittableList world = HittableList();
	
	world.add(make_shared<Sphere>(
		Vector3(0, -1000, 0), 1000, make_shared<Lambertian>(Vector3(.3, .5, .5))));

	for (int a = -11; a < 11; a++)
	{
		for (int b = -11; b < 11; b++)
		{
			double chooseMat = ffrnd();
			Vector3 center = Vector3(a + .9 * ffrnd(), .2, b + .9 * ffrnd());
			if ((center - Vector3(4, .2, 0)).Length() > .9)
			{
				if (chooseMat < .8)
				{
					//diffuse
					Vector3 albedo = mult(Vector3::Random(), Vector3::Random());
					world.add(
						make_shared<Sphere>(center, .2, make_shared<Lambertian>(albedo))
					);
				}
				else if (chooseMat < .95)
				{
					// metal
					Vector3 albedo = Vector3::Random(.5, 1);
					double fuzz = ffrnd(0, .5);
					world.add(
						make_shared<Sphere>(center, .2, make_shared<Metal>(albedo, fuzz))
					);
				}
				else
				{
					// glass
					world.add(make_shared<Sphere>(center, .2, make_shared<Dielectric>(1.5)));
				}
			}
		}
	}

	

	world.add(make_shared<Sphere>(Vector3(0, 1, 0), 1.0, make_shared<Dielectric>(1.5)));

	world.add(
		make_shared<Sphere>(Vector3(-4, 1, 0), 1.0, make_shared<Lambertian>(Vector3(0.4, 0.2, 0.1))));

	world.add(
		make_shared<Sphere>(Vector3(4, 1, 0), 1.0, make_shared<Metal>(Vector3(0.7, 0.6, 0.5), 0.0)));

	return world;
}

void threadAction(Camera& cam, const Hittable& world, Vector3** colors, int offset)
{
	for (int N = offset; N < WIDTH * HEIGHT; N += NUMBER_OF_THREADS)
	{
		int i = N % WIDTH;
		int j = (N / WIDTH) % HEIGHT;
		Vector3 color;
		// SAMPLING LINEAR
		for (int xs = 1; xs <= SAMPLES_PER_PIXEL_SQRT; ++xs)
		{
			for (int ys = 1; ys <= SAMPLES_PER_PIXEL_SQRT; ++ys)
			{
				double u = (double(i) + double(xs) / (SAMPLES_PER_PIXEL_SQRT + 2)) / double(WIDTH);
				double v = (double(j) + double(ys) / (SAMPLES_PER_PIXEL_SQRT + 2)) / double(HEIGHT);
				Ray r = cam.GetRay(u, v);
				color += RayColor(r, world, MAX_DEPTH);
			}
		}
		colors[j][i] = color;
	}
}

// Color calculation
Vector3 RayColor(const Ray& r, const Hittable& world, int depth)
{
	if (depth <= 0)
		return Vector3();

	// Checking for collisoin with world object
	HitRecord rec;
	if (world.Hit(r, 0.001, infinity, rec))
	{
		Vector3 target;

		Ray scattered = Ray(Vector3(), Vector3());
		Vector3 attenuation;
		if (rec.MatPtr->Scatter(r, rec, attenuation, scattered))
			return mult(attenuation, RayColor(scattered, world, depth - 1));

		return Vector3(0, 0, 0);
	}

	// otherwise returning sky background
	Vector3 unitDir = r.Direction.Normalized();
	double t = .5 * (unitDir.y + 1.0);
	return (1.0 - t) * Vector3(1, 1, 1) + t * Vector3(0.5, 0.7, 1.0);
}

int main(void)
{
	ofstream file;
	file.open(FILEPATH);

	file << "P3\n" << WIDTH << ' ' << HEIGHT << "\n255\n";

	// Creating the World
	HittableList world = RandomScene();

	Vector3 lookfrom = Vector3(13, 2, 3);
	Vector3 lookat = Vector3(0, 0, 0);
	Vector3 vup = Vector3(0, 1, 0);
	double dist_to_focus = 10.0;
	double aperture = .1;

	Camera cam = Camera(lookfrom, lookat, vup, 20, ASPECT_RATIO, aperture, dist_to_focus);

	// Creating colors array
	Vector3** colors = (Vector3**)malloc(sizeof(Vector3*) * WIDTH);
	for (int i = 0; i < WIDTH; ++i)
	{
		colors[i] = (Vector3*)malloc(sizeof(Vector3*) * HEIGHT);
	}

	// Creating array of all threads
	thread threads[NUMBER_OF_THREADS];
	for (int i = 0; i < NUMBER_OF_THREADS; i++)
	{
		threads[i] = thread(threadAction, cam, world, colors, i);
	}

	for (int i = 0; i < NUMBER_OF_THREADS; i++)
	{
		threads[i].join();
	}

	/*
	int currentThread = 0;
	for (int j = HEIGHT - 1; j >= 0; --j)
	{
		if (j % 1 == 0)
			std::cout << double(HEIGHT - j) / HEIGHT << "   PERCENT" << std::endl;
		for (int i = 0; i < WIDTH; ++i)
		{
			Vector3 color = Vector3();

			// Await thread and then call again
			if (!threads[currentThread].joinable())
			{
				cout << "fiuawfbaiwoubf " << i << " "  << currentThread << " "  << j << endl;
				cin.get();
			}
			threads[currentThread].join();
			threads[currentThread] = thread(threadAction, cam, world, colors, i, j);

			++currentThread;
			if (currentThread >= NUMBER_OF_THREADS)
				currentThread = 0;
		}
	}
	*/

	// Drawing the picture
	for (int j = HEIGHT - 1; j >= 0; --j)
	{
		for (int i = 0; i < WIDTH; ++i)
		{
			colors[i][j].WriteColor(file, SAMPLES_PER_PIXEL_SQRT * SAMPLES_PER_PIXEL_SQRT, GAMMA_VALUE);
		}
	}

	for (int i = 0; i < WIDTH; ++i)
	{
		free(colors[i]);
	}
	free(colors);

	file.close();
	return 0;
}

/*
__global__
void CollideRay(RayCollisionObject* solids, int N, RayCollisionObject* lights, int M)
{
	return;
}

__global__
void GetColorOfRay(Ray r, RayCollisionObject* solids, int N, RayCollisionObject* lights, int M)
{
	int nearestSolid = -1;
	double minSolid = -1;
	double tmp = 0;
	for (int i = 0; i < N; i++)
	{
		if (solids[i].Coliding(r)) {
			if (minSolid == -1) {
				minSolid = solids[i].Distance(r.getOrigin());
				nearestSolid = i;
			}
			else
			{
				tmp = solids[i].Distance(r.getOrigin());
				minSolid = tmp < minSolid ? tmp : minSolid;
				if (tmp == minSolid)
					nearestSolid = i;
			}
		}
	}
	
	int nearesLight = -1;
	double minLight = -1;
	for (int i = 0; i < M; i++)
	{
		if (lights[i].Coliding(r))
		{
			if (minLight == -1) {
				minLight = solids[i].Distance(r.getOrigin());
				nearesLight = i;
			}
			else
			{
				tmp = solids[i].Distance(r.getOrigin());
				minLight = tmp < minSolid ? tmp : minSolid;
				if (tmp == minLight)
					nearesLight = i;
			}
		}
	}
	
	if (nearesLight != -1 && nearestSolid == -1)
		return lights[nearesLight].GetColor();
	else if (nearesLight != -1 && nearestSolid != -1 && minLight < minSolid)
		return lights[nearesLight].GetColor();
	if (nearestSolid == -1)
		return { 0, 0, 0 };
	

	int K = 0;
	Ray* rays = solids[nearestSolid].GetConsecutiveRays(r, &K);
	if (K == 0)
		return { 0, 0, 0 };

	Color* colors = nullptr;

	hipMalloc(&colors, K * sizeof(Color));
	hipFree(&colors);
	
}
*/
